
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
//Single-Threaded Program

void random(int *array, int SIZE) {
    for (int i = 0; i < SIZE; i++) {
        array[i] = rand();
    }
}

int main() {
    int SIZE = 536870912;
    int *x = (int*)malloc(SIZE * sizeof(int));
    int *y = (int*)malloc(SIZE * sizeof(int));
    int *z = (int*)malloc(SIZE * sizeof(int));

    random(x, SIZE);
    random(y, SIZE);

    int c = rand() % 100;
    printf("%d\n", c);
    printf("%d\n", x[3]);
    printf("Hello World!TEST\n");

    for (int i = 0; i < SIZE; i++) {
        z[i] = x[i] * c + y[i];
    }

    //Free Memory
    free(x);
    free(y);
    free(z);

    return 0;
}

