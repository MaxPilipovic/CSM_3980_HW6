
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
//Single-Threaded Program

void random(int *array, int SIZE) {
    for (int i = 0; i < SIZE; i++) {
        array[i] = rand();
    }
}

int main() {
    int SIZE = 536870912;
    int x[SIZE];
    int y[SIZE];
    int z[SIZE];

    random(x, SIZE);
    random(y, SIZE);

    //Number between 1 and 100
    int c = rand() % 100 + 1;

    for (int j = 0; j < 0xFFFFFFF; j++) {
        int ticks = clock();

        for (int i = 0; i < SIZE; i++) {
            z[i] = x[i] * c + y[i];
        }
        printf("%d\n", c);
        printf("%d\n", x[3]);
        printf("Hello World!TEST\n");
        printf("\n");
        printf("%f\n", (float)ticks / CLOCKS_PER_SEC);
        break;
    }


    //Free Memory
    free(x);
    free(y);
    free(z);

    return 0;
}

