
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
//Single-Threaded Program
void random(int *array, int SIZE) {
    for (int i = 0; i < SIZE; i++) {
        array[i] = rand();
    }
}

int main() {
    //here
    srand(time(NULL));
    int SIZE = 536870912; //2GB
    int *x = (int*)malloc(SIZE * sizeof(int));
    int *y = (int*)malloc(SIZE * sizeof(int));
    int *z = (int*)malloc(SIZE * sizeof(int));

    random(x, SIZE);
    random(y, SIZE);

    //Number between 1 and 100
    int c = rand() % 100 + 1;
    clock_t start_t, end_t;
    double total_t;

    start_t = clock();
    for (int i = 0; i < SIZE; i++) {
        z[i] = x[i] * c + y[i];
    }
    end_t = clock();
    total_t = (double)(end_t - start_t) / CLOCKS_PER_SEC;
    printf("%f\n", total_t);

    printf("%d\n", c);
    printf("%d\n", x[3]);
    printf("Hello World!TEST\n");
    printf("\n");

    //Free Memory
    free(x);
    free(y);
    free(z);

    return 0;
}

