#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
//Single-Threaded Program
void random(int *array, int SIZE) {
    for (int i = 0; i < SIZE; i++) {
        array[i] = rand();
    }
}

int main() {
    //here
    srand(time(NULL));
    //268435456
    int SIZE = 536870912;
    int *x = (int*)malloc(SIZE * sizeof(int));
    int *y = (int*)malloc(SIZE * sizeof(int));
    int *z = (int*)malloc(SIZE * sizeof(int));

    random(x, SIZE);
    random(y, SIZE);

    //Number between 1 and 100
    time_t start = time(NULL);

    start_t = clock();
    for (int i = 0; i < SIZE; i++) {
        z[i] = x[i] * c + y[i];
    }
    time_t end = time(NULL);
    printf("%d\n", difftime(end, start));

    printf("%d\n", c);
    printf("%d\n", x[3]);
    printf("Hello World!TEST\n");
    printf("\n");

    //Free Memory
    free(x);
    free(y);
    free(z);

    return 0;
}

