#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
//Adjacent Multi-Threaded CUDA
void random(int *array, int size_t) {
    for (int i = 0; i < size_t; i++) {
        array[i] = rand();
    }
}

__global__ void vecadd_kernel(int* x, int* y, int* z, int c, size_t) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    //Works on different array elements seperated by stride
    for (int j = i; j < size_t; j += stride) {
        z[j] = c * x[j] + y[j];
    }
}

void vecadd(int* x, int* y, int* z, int c, size_t) {
    //Allocate GPU memory
    int *x_d, *y_d, *z_d;

    hipMalloc((void**) &x_d, size_t*sizeof(int));
    hipMalloc((void**) &y_d, size_t*sizeof(int));
    hipMalloc((void**) &z_d, size_t*sizeof(int));

    //Copy data to GPU memory
    hipMemcpy(x_d, x, size_t*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, size_t*sizeof(int), hipMemcpyHostToDevice);

    //Perform computation on GPU
    int numThreadsPerBlock = 512;
    int numBlocks = (SIZE + numThreadsPerBlock - 1) / numThreadsPerBlock;

    //Start time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    float time;

    vecadd_kernel<<<numBlocks, numThreadsPerBlock>>>(x_d, y_d, z_d, c, size_t);

    //End time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("%f\n", time);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //Synchronize
    hipDeviceSynchronize();

    //Copy data from GPU memory
    hipMemcpy(z, z_d, SIZE *sizeof(int), hipMemcpyDeviceToHost);

    //Deallocate GPU memory
    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
}

int main() {
    srand(time(NULL));
    //268435456
    //805306368
    //1073741824
    size_t = 1610612736;
    int *x = (int*)malloc(size_t * sizeof(int));
    int *y = (int*)malloc(SIZE * sizeof(int));
    int *z = (int*)malloc(SIZE * sizeof(int));

    random(x, size_t);
    random(y, size_t);

    //Number between 1 and 100
    int c = rand() % 100 + 1;

    //Send it
    vecadd(x, y, z, c, size_t);

    free(x);
    free(y);
    free(z);

    return 0;
}