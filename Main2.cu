
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
//Adjacent Multi-Threaded CUDA
void random(int *array, int SIZE) {
    for (int i = 0; i < SIZE; i++) {
        array[i] = rand();
    }
}

__global__ void vecadd_kernel(int* x, int* y, int* z, int c, int n) {
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    //Works on different array elements seperated by stride
    for (int j = i; j < n; j += stride) {
        z[j] = c * x[j] + y[j];
    }
}

void vecadd(int* x, int* y, int* z, int c, int SIZE) {
    // Allocate GPU memory
    int *x_d, *y_d, *z_d;

    hipMalloc((void**) &x_d, SIZE*sizeof(int));
    hipMalloc((void**) &y_d, SIZE*sizeof(int));
    hipMalloc((void**) &z_d, SIZE*sizeof(int));
    // Copy data to GPU memory
    hipMemcpy(x_d, x, SIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, SIZE*sizeof(int), hipMemcpyHostToDevice);

    // Perform computation on GPU
    int numThreadsPerBlock = 512;
    int numBlocks = (SIZE + numThreadsPerBlock - 1) / numThreadsPerBlock;
    vecadd_kernel<<<numBlocks, numThreadsPerBlock>>>(x_d, y_d, z_d, c, SIZE);

    //Synchronize
    hipDeviceSynchronize();

    // Copy data from GPU memory
    hipMemcpy(z, z_d, SIZE *sizeof(int), hipMemcpyDeviceToHost);

    // Deallocate GPU memory
    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
}

int main() {
    srand(time(NULL));
    //268435456
    int SIZE = 134217728;
    int *x = (int*)malloc(SIZE * sizeof(int));
    int *y = (int*)malloc(SIZE * sizeof(int));
    int *z = (int*)malloc(SIZE * sizeof(int));

    random(x, SIZE);
    random(y, SIZE);

    //Number between 1 and 100
    int c = rand() % 100 + 1;

    clock_t start_t, end_t;
    double total_t;
    start_t = clock();

    //Send it
    vecadd(x, y, z, c, SIZE);

    end_t = clock();
    total_t = (double)(end_t - start_t) / CLOCKS_PER_SEC;
    printf("%f\n", total_t);


    free(x);
    free(y);
    free(z);

    return 0;
}