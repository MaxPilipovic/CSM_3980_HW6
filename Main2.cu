#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
//Adjacent Multi-Threaded CUDA
void random(float *array, float SIZE) {
    for (float i = 0; i < SIZE; i++) {
        array[i] = rand();
    }
}

__global__ void vecadd_kernel(float* x, float* y, float* z, float c, float n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    //Works on different array elements seperated by stride
    for (int j = i; j < n; j += stride) {
        z[j] = c * x[j] + y[j];
    }
}

void vecadd(float* x, float* y, float* z, float c, float SIZE) {
    //Allocate GPU memory
    float *x_d, *y_d, *z_d;

    hipMalloc((void**) &x_d, SIZE*sizeof(float));
    hipMalloc((void**) &y_d, SIZE*sizeof(float));
    hipMalloc((void**) &z_d, SIZE*sizeof(float));

    //Copy data to GPU memory
    hipMemcpy(x_d, x, SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, SIZE*sizeof(float), hipMemcpyHostToDevice);

    //Start time
    //clock_t start_t, end_t;
    //double total_t;
    //start_t = clock();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    float time;

    //Perform computation on GPU
    int numThreadsPerBlock = 512;
    int numBlocks = (SIZE + numThreadsPerBlock - 1) / numThreadsPerBlock;
    vecadd_kernel<<<numBlocks, numThreadsPerBlock>>>(x_d, y_d, z_d, c, SIZE);

    //end_t = clock();
    //total_t = (double)(end_t - start_t) / CLOCKS_PER_SEC;
    //printf("%f\n", total_t);

    //End time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("%f\n", time);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //Synchronize
    hipDeviceSynchronize();

    //Copy data from GPU memory
    hipMemcpy(z, z_d, SIZE *sizeof(float), hipMemcpyDeviceToHost);

    //Deallocate GPU memory
    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
}

int main() {
    srand(time(NULL));
    //268435456
    //805306368
    //1073741824
    float SIZE = 1610612736;
    float *x = (float*)malloc(SIZE * sizeof(float));
    float *y = (float*)malloc(SIZE * sizeof(float));
    float *z = (float*)malloc(SIZE * sizeof(float));

    random(x, SIZE);
    random(y, SIZE);

    //Number between 1 and 100
    float c = rand() % 100 + 1;

    //Send it
    vecadd(x, y, z, c, SIZE);

    free(x);
    free(y);
    free(z);

    return 0;
}